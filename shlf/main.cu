
#include <hip/hip_runtime.h>
#include <stdio.h>

typedef struct __align__(4)
{
	double b;
    float a;
} point;

__global__ void testKernel(point *p)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    p[i].a = 1.1;
    p[i].b = 2.2;
}

int main(void)
{
        // set number of points 
    int numPoints    = 16,
        gpuBlockSize = 4,
        pointSize    = sizeof(point),
        numBytes     = numPoints * pointSize,
        gpuGridSize  = numPoints / gpuBlockSize;
	printf("%d\n",pointSize);
        // allocate memory
    point *cpuPointArray = new point[numPoints],
          *gpuPointArray;
    cpuPointArray = (point*)malloc(numBytes);
    hipMalloc((void**)&gpuPointArray, numBytes);

        // launch kernel
    testKernel<<<gpuGridSize,gpuBlockSize>>>(gpuPointArray);

        // retrieve the results
    hipMemcpy(cpuPointArray, gpuPointArray, numBytes, hipMemcpyDeviceToHost);
    printf("testKernel results:\n");
    for(int i = 0; i < numPoints; ++i)
    {
        printf("point.a: %f, point.b: %f\n",cpuPointArray[i].a,cpuPointArray[i].b);
    }

        // deallocate memory
    free(cpuPointArray);
    hipFree(gpuPointArray);

    return 0;
}

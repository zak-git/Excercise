
#include <hip/hip_runtime.h>
#include <stdio.h>
int host_x[4] = {1, 2, 3, 4};
//__constant__ int dev_x[4];
__global__ void kernel(int *d_var) {
	d_var[threadIdx.x] += 10;
	float2 f = make_float2(1.f,0.f); 
}
__global__ void init(int *d_var) { d_var[threadIdx.x] = threadIdx.x + 1; }
int main() 
{
	int data_size = 4 * sizeof(int);
	int *address;
    //cudaMalloc((void**) &dev_x, data_size);
    //cudaMemcpyToSymbol(dev_x, host_x, data_size,0, cudaMemcpyHostToDevice);
    //cudaGetSymbolAddress((void**)&address, dev_x);
	hipMalloc((void**) &address, data_size);
	//cudaMemcpy(address, host_x, data_size, cudaMemcpyHostToDevice);
    init<<<1,4>>>(address);
    kernel<<<1,4>>>(address);
	hipDeviceSynchronize();

	hipMemcpy(host_x, address, data_size, hipMemcpyDeviceToHost);

	for (int i=0; i< 4; i++){
		printf("%d\n", host_x[i]);
	}
	return 0;
}
